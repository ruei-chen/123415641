#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <cstdlib>
#include <cstring>
#include <chrono>

using namespace std;
using namespace std::chrono;

// 定義 Point 類別
class Point
{
public:
    int pointId;
    int clusterId;
    int dimensions;
    double *values;

    // 預設建構函數
    Point(int id, double *vals, int dim) : pointId(id), clusterId(-1), dimensions(dim)
    {
        values = new double[dim];
        memcpy(values, vals, dim * sizeof(double));
    }

    // 深拷貝建構函數
    Point(const Point &other) : pointId(other.pointId), clusterId(other.clusterId), dimensions(other.dimensions)
    {
        values = new double[dimensions];
        memcpy(values, other.values, dimensions * sizeof(double));
    }

    // 深拷貝賦值運算符
    Point &operator=(const Point &other)
    {
        if (this == &other)
            return *this; // 避免自我賦值
        delete[] values;  // 釋放原來的記憶體
        pointId = other.pointId;
        clusterId = other.clusterId;
        dimensions = other.dimensions;
        values = new double[dimensions];
        memcpy(values, other.values, dimensions * sizeof(double));
        return *this;
    }

    // 解構函數
    ~Point()
    {
        delete[] values;
    }
};

// 自定義 atomicAddDouble，用於支援 double 原子加法
__device__ double atomicAddDouble(double *address, double val)
{
    unsigned long long int *address_as_ull = (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

// CUDA 核心函數：計算最近叢集
// points 每個點的座標  centroid 每個群的中心  clusterassignment 每個點哪一群   total_points 點的總數
__global__ void assignClusters(double *points, double *centroids, int *clusterAssignments,
                               int total_points, int K, int dimensions)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_points)
        return;

    int nearestClusterId = -1;
    double minDist = 10000;

    for (int k = 0; k < K; ++k)
    {
        double sum = 0.0;
        for (int d = 0; d < dimensions; ++d)
        {
            double diff = points[idx * dimensions + d] - centroids[k * dimensions + d];
            sum += diff * diff;
        }
        double distance = sqrt(sum);
        if (distance < minDist)
        {
            minDist = distance;
            nearestClusterId = k;
        }
    }
    clusterAssignments[idx] = nearestClusterId;
}

// CUDA 核心函數：計算新的叢集中心
// points 每個點的座標  centroid 每個群的中心  clusterassignment 每個點哪一群   total_points 點的總數
__global__ void computeNewCentroids(double *points, double *centroids, int *clusterAssignments,
                                    int *clusterSizes, int total_points, int K, int dimensions)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_points)
        return;

    int clusterId = clusterAssignments[idx];
    for (int d = 0; d < dimensions; ++d)
    {
        atomicAddDouble(&centroids[clusterId * dimensions + d], points[idx * dimensions + d]);
    }
    atomicAdd(&clusterSizes[clusterId], 1);
}

// 定義 KMeans 類別
class KMeans
{
private:
    int K, dimensions, total_points;
    string output_dir;

public:
    KMeans(int K, string output_dir) : K(K), output_dir(output_dir) {}

    void run(vector<Point> &all_points)
    {
        total_points = all_points.size();
        dimensions = all_points[0].dimensions;

        // 主機記憶體分配
        double *host_points = new double[total_points * dimensions];
        double *host_centroids = new double[K * dimensions];
        int *host_clusterAssignments = new int[total_points];
        int *host_clusterSizes = new int[K]();

        // 初始化點座標與叢集中心
        for (int i = 0; i < total_points; ++i)
        {
            memcpy(&host_points[i * dimensions], all_points[i].values, dimensions * sizeof(double));
        }

        for (int i = 0; i < K; ++i)
        {
            int randomPointIndex = rand() % total_points;
            memcpy(&host_centroids[i * dimensions], all_points[randomPointIndex].values, dimensions * sizeof(double));
        }

        // 設備記憶體分配
        double *device_points, *device_centroids;
        int *device_clusterAssignments, *device_clusterSizes;
        hipMalloc(&device_points, total_points * dimensions * sizeof(double));
        hipMalloc(&device_centroids, K * dimensions * sizeof(double));
        hipMalloc(&device_clusterAssignments, total_points * sizeof(int));
        hipMalloc(&device_clusterSizes, K * sizeof(int));

        // 主機資料拷貝到設備
        hipMemcpy(device_points, host_points, total_points * dimensions * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(device_centroids, host_centroids, K * dimensions * sizeof(double), hipMemcpyHostToDevice);

        const int threadsPerBlock = 256;
        const int blocksPerGrid = (total_points + threadsPerBlock - 1) / threadsPerBlock;

        for (int iter = 0; iter < 100; ++iter)
        {
            // 清空叢集大小
            // cout << "Iter - " << iter + 1 << "/ 100" << endl;
            hipMemset(device_clusterSizes, 0, K * sizeof(int));
            hipMemset(device_clusterAssignments, -1, total_points * sizeof(int));

            // 分配最近叢集
            assignClusters<<<blocksPerGrid, threadsPerBlock>>>(device_points, device_centroids, device_clusterAssignments,
                                                               total_points, K, dimensions);
            hipDeviceSynchronize();

            // 計算新叢集中心
            // points 每個點的座標   clusterassignment 每個點哪一群   total_points 點的總數
            hipMemset(device_centroids, 0, K * dimensions * sizeof(double));
            computeNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(device_points, device_centroids, device_clusterAssignments,
                                                                    device_clusterSizes, total_points, K, dimensions);
            hipDeviceSynchronize();

            // 更新叢集中心平均值
            hipMemcpy(host_centroids, device_centroids, K * dimensions * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(host_clusterSizes, device_clusterSizes, K * sizeof(int), hipMemcpyDeviceToHost);

            for (int k = 0; k < K; ++k)
            {
                for (int d = 0; d < dimensions; ++d)
                {
                    if (host_clusterSizes[k] > 0)
                    {
                        // centroid 每個群的中心  clustersize 每個群的點數
                        host_centroids[k * dimensions + d] /= host_clusterSizes[k];
                    }
                }
            }

            // 拷貝更新後的叢集中心到設備
            hipMemcpy(device_centroids, host_centroids, K * dimensions * sizeof(double), hipMemcpyHostToDevice);
        }

        // 複製結果到主機
        hipMemcpy(host_clusterAssignments, device_clusterAssignments, total_points * sizeof(int), hipMemcpyDeviceToHost);

        // 儲存結果
        ofstream centroidsFile(output_dir + "/centroids.txt");
        ofstream pointsFile(output_dir + "/points.txt");
        for (int k = 0; k < K; ++k)
        {
            for (int d = 0; d < dimensions; ++d)
            {
                centroidsFile << host_centroids[k * dimensions + d];
                if (d < dimensions - 1)
                {
                    centroidsFile << " "; // 座標之間用空格分隔
                }
            }
            centroidsFile << endl; // 每個重心座標換行
        }
        centroidsFile.close();

        // 儲存點所屬叢集編號
        for (int i = 0; i < total_points; ++i)
        {
            pointsFile << (host_clusterAssignments[i] + 1) << endl; // 群編號調整為 1-K
        }
        pointsFile.close();
        // 釋放設備記憶體
        hipFree(device_points);
        hipFree(device_centroids);
        hipFree(device_clusterAssignments);
        hipFree(device_clusterSizes);

        // 釋放主機記憶體
        delete[] host_points;
        delete[] host_centroids;
        delete[] host_clusterAssignments;
        delete[] host_clusterSizes;
    }
};

int main(int argc, char **argv)
{
    if (argc != 4)
    {
        cout << "Usage: ./final <INPUT> <K> <OUTPUT_DIR>" << endl;
        return 1;
    }

    string input_file = argv[1];
    int K = atoi(argv[2]);
    string output_dir = argv[3];

    ifstream infile(input_file);
    if (!infile.is_open())
    {
        cout << "Error: Unable to open file " << input_file << endl;
        return 1;
    }

    vector<Point> all_points;
    string line;
    int pointId = 0;

    // 跳過表頭
    getline(infile, line);

    while (getline(infile, line))
    {
        stringstream ss(line);
        string cell;
        vector<double> values;

        int columnIdx = 0;
        while (getline(ss, cell, '\t'))
        { // 使用 '\t' 作為分隔符
            columnIdx++;

            // 選擇只處理數值列，例如第 4 到第 10 列
            if (columnIdx >= 5 && columnIdx <= 20 && columnIdx != 8)
            {
                try
                {
                    values.push_back(stod(cell));
                }
                catch (const invalid_argument &e)
                {
                    // 忽略非數值字段
                    continue;
                }
            }
        }

        if (!values.empty())
        {
            all_points.emplace_back(pointId++, values.data(), values.size());
        }
    }

    infile.close();

    if (all_points.empty())
    {
        cerr << "Error: No valid points parsed from the input file. Please check the file format and column indices." << endl;
        return 1;
    }

    cout << "\nData fetched successfully!" << endl
         << endl;

    auto start_time = high_resolution_clock::now();

    // 執行 KMeans
    for (int i = 3; i <= K; i++)
    {
        KMeans kmeans(K, output_dir);
        kmeans.run(all_points);
    }

    auto end_time = high_resolution_clock::now(); // End timing for iteration
    auto iter_duration = duration_cast<milliseconds>(end_time - start_time);
    cout << "Program completed in " << iter_duration.count() << " ms" << endl;

    return 0;
}
